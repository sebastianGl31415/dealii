// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check that we detect creating ArrayView objects with the wrong memory space.

#include <deal.II/base/array_view.h>

#include "../tests.h"

int
main(int argc, char **argv)
{
  deal_II_exceptions::disable_abort_on_exception();

  initlog();

  init_cuda();

  std::vector<unsigned int>                                 dummy_host(2);
  std::unique_ptr<unsigned int[], void (*)(unsigned int *)> dummy_cuda(
    Utilities::CUDA::allocate_device_data<unsigned int>(2),
    Utilities::CUDA::delete_device_data<unsigned int>);

  deallog << "Testing host ArrayView with host memory" << std::endl;
  ArrayView<unsigned int, MemorySpace::Host> view_1(dummy_host);

  deallog << "Testing device ArrayView with host memory" << std::endl;
  try
    {
      ArrayView<unsigned int, MemorySpace::CUDA> view_2(dummy_host);
    }
  catch (const ExceptionBase &exc)
    {
      deallog << exc.what() << std::endl;
    }

  deallog << "Testing host ArrayView with device memory" << std::endl;
  try
    {
      ArrayView<unsigned int, MemorySpace::Host> view_3(dummy_cuda.get(), 2);
    }
  catch (const ExceptionBase &exc)
    {
      deallog << exc.what() << std::endl;
    }

  deallog << "Testing device ArrayView with device memory" << std::endl;
  ArrayView<unsigned int, MemorySpace::CUDA> view_4(dummy_cuda.get(), 2);

  deallog << "Testing host ArrayView to a nullptr with length 0" << std::endl;
  ArrayView<unsigned int, MemorySpace::Host> view_5(nullptr, 0);

  deallog << "Testing device ArrayView to a nullptr with length 0" << std::endl;
  ArrayView<unsigned int, MemorySpace::CUDA> view_6(nullptr, 0);

  return 0;
}
